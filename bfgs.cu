#include <iostream>
#include <vector>
#include <fstream>
#include <limits> // Per std::numeric_limits
#include <cmath>  // Per std::isnan
#include <hip/hip_runtime.h>

#define MAX_FRONTIER_SIZE 128
struct Pair {
    int first;
    int second;
};
#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void csr_to_dense(std::ifstream &file, std::vector<std::vector<float>> &denseMatrix) {
    int numRows, numCols, numValues;
    file >> numRows >> numCols >> numValues;

    // Inizializza la matrice con NaN
    denseMatrix = std::vector<std::vector<float>>(numRows, std::vector<float>(numCols, std::numeric_limits<float>::quiet_NaN()));

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        row--; // Convert to 0-based index
        col--; // Convert to 0-based index
        denseMatrix[row][col] = value;
    }
}

void generate_matrix_file(const std::vector<std::vector<float>> &matrix, const std::string &filename) {
    std::ofstream matrixFile(filename);
    if (!matrixFile.is_open()) {
        std::cerr << "Cannot open matrix file!\n";
        return;
    }

    matrixFile << "Dense Matrix:" << std::endl;
    for (const auto &row : matrix) {
        for (const auto &val : row) {
            if (std::isnan(val)) {
                matrixFile << "- ";
            } else {
                matrixFile << val << " ";
            }
        }
        matrixFile << std::endl;
    }

    matrixFile.close();
}

__global__ void BFS_parallel(int source, Pair* currentFrontier, int* currentFrontierSize, Pair* nextFrontier, int* nextFrontierSize, float* denseMatrix, int numRows, int numCols, int* distances) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= *currentFrontierSize) return;
    int d=0;
    while (true) {
        Pair node = currentFrontier[tid];
        bool found = false;
        // Controllo per i < j lungo la colonna
        if (node.first < node.second) {
            for (int j = 0; j < numCols; ++j) {
                if (node.first < j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        } else if (node.first > node.second) {
            // Controllo per i > j lungo la riga
            for (int j = 0; j < numRows; ++j) {
                if (node.first > j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        } else {
            // Controllo per i == j lungo la diagonale
            for (int j = 0; j < numCols; ++j) {
                if (node.first == j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    distances[d] = node.first; // Aggiorna la distanza
                    d++;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        }

        if (found && node.first == source && node.second == source) {
            break; // Interrompe il ciclo se il nodo source è stato trovato
        }

        __syncthreads();

        if (tid == 0) {
            *currentFrontierSize = *nextFrontierSize;
            *nextFrontierSize = 0;
            Pair* temp = currentFrontier;
            currentFrontier = nextFrontier;
            nextFrontier = temp;
        }

        __syncthreads();

        if (*currentFrontierSize == 0) {
            break; // Interrompe il ciclo se non ci sono più nodi da esplorare
        }
    }
}

void generate_distance_file(const std::vector<int> &distances, int source, int numRows, int numCols, const std::string &filename) {
    std::ofstream distanceFile(filename);
    if (!distanceFile.is_open()) {
        std::cerr << "Cannot open distance file!\n";
        return;
    }

    distanceFile << "Distances:" << std::endl;
    for (int i = 0; i < source; ++i) {
            int dist=source-i-1;
            distanceFile << "Node (" << distances[i] << ", " << distances[i] << "): " << dist;
            if (i != (source)) {
                distanceFile <<std::endl;
            } else {
                distanceFile << " (source)" << std::endl;
           }
           
    }
    distanceFile.close();
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cout << "Usage: ./exec matrix_file source\n";
        return 0;
    }

    std::string filename = argv[1];
    int source = atoi(argv[2]);

    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        return 1;
    }

    std::vector<std::vector<float>> denseMatrix;
    csr_to_dense(file, denseMatrix);
    std::cout << "Matrice formata\n";
    file.close();

    int numRows = denseMatrix.size();
    int numCols = denseMatrix[0].size();
    float* d_denseMatrix;
    hipMalloc(&d_denseMatrix, numRows * numCols * sizeof(float));
    hipMemcpy(d_denseMatrix, denseMatrix.data(), numRows * numCols * sizeof(float), hipMemcpyHostToDevice);

    // Inizializza currentFrontier con una singola coppia
    std::vector<Pair> hostCurrentFrontier = {{0, 0}};
    Pair* d_currentFrontier;
    int* d_currentFrontierSize;
    hipMalloc(&d_currentFrontier, hostCurrentFrontier.size() * sizeof(Pair));
    hipMalloc(&d_currentFrontierSize, sizeof(int));
    hipMemcpy(d_currentFrontier, hostCurrentFrontier.data(), hostCurrentFrontier.size() * sizeof(Pair), hipMemcpyHostToDevice);
    int currentFrontierSize = hostCurrentFrontier.size();
    hipMemcpy(d_currentFrontierSize, &currentFrontierSize, sizeof(int), hipMemcpyHostToDevice);

    // Inizializza nextFrontier vuoto
    Pair* d_nextFrontier;
    int* d_nextFrontierSize;
    hipMalloc(&d_nextFrontier, numRows * numCols * sizeof(Pair)); // Assumiamo che la dimensione massima sia numRows * numCols
    hipMalloc(&d_nextFrontierSize, sizeof(int));
    int nextFrontierSize = 0;
    hipMemcpy(d_nextFrontierSize, &nextFrontierSize, sizeof(int), hipMemcpyHostToDevice);

    // Inizializza il vettore delle distanze
    std::vector<int> hostDistances(numRows, -1 ); // Inizializza tutte le distanze a -1
    hostDistances[(source - 1)] = source; // La distanza del nodo source è 0
    int* d_distances;
    hipMalloc(&d_distances, sizeof(int));
    hipMemcpy(d_distances, hostDistances.data(), sizeof(int), hipMemcpyHostToDevice);
    
    // Variabili per la temporizzazione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Avvia il timer
    hipEventRecord(start);

    int blockSize = 256;
    int numBlocks = (currentFrontierSize + blockSize - 1) / blockSize;
    BFS_parallel<<<numBlocks, blockSize>>>(source, d_currentFrontier, d_currentFrontierSize, d_nextFrontier, d_nextFrontierSize, d_denseMatrix, numRows, numCols, d_distances);

    // Ferma il timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcola il tempo di esecuzione
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Tempo di esecuzione del kernel: " << milliseconds << " ms" << std::endl;
    // Copia i risultati dalla GPU all'host
    hipMemcpy(hostDistances.data(), d_distances, sizeof(int), hipMemcpyDeviceToHost);

    // Genera il file delle distanze
    generate_distance_file(hostDistances, source, numRows, numCols, "distances.txt");

    hipFree(d_denseMatrix);
    hipFree(d_currentFrontier);
    hipFree(d_currentFrontierSize);
    hipFree(d_nextFrontier);
    hipFree(d_nextFrontierSize);
    hipFree(d_distances);
    hipDeviceReset();

    return 0;
}
