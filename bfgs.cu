#include <iostream>
#include <vector>
#include <fstream>
#include <limits> // Per std::numeric_limits
#include <cmath>  // Per std::isnan
#include <hip/hip_runtime.h>

#define MAX_FRONTIER_SIZE 128
struct Pair {
    int first;
    int second;
};
#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void csr_to_dense(std::ifstream &file, std::vector<std::vector<float>> &denseMatrix) {
    int numRows, numCols, numValues;
    file >> numRows >> numCols >> numValues;

    // Inizializza la matrice con NaN
    denseMatrix = std::vector<std::vector<float>>(numRows, std::vector<float>(numCols, std::numeric_limits<float>::quiet_NaN()));

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        row--; // Convert to 0-based index
        col--; // Convert to 0-based index
        denseMatrix[row][col] = value;
    }
}

void generate_matrix_file(const std::vector<std::vector<float>> &matrix, const std::string &filename) {
    std::ofstream matrixFile(filename);
    if (!matrixFile.is_open()) {
        std::cerr << "Cannot open matrix file!\n";
        return;
    }

    matrixFile << "Dense Matrix:" << std::endl;
    for (const auto &row : matrix) {
        for (const auto &val : row) {
            if (std::isnan(val)) {
                matrixFile << "- ";
            } else {
                matrixFile << val << " ";
            }
        }
        matrixFile << std::endl;
    }

    matrixFile.close();
}

__global__ void BFS_parallel(int source, Pair* currentFrontier, int* currentFrontierSize, Pair* nextFrontier, int* nextFrontierSize, float* denseMatrix, int numRows, int numCols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= *currentFrontierSize) return;

    while (true) {
        Pair node = currentFrontier[tid];
        bool found = false;

        // Controllo per i < j
        if (node.first < node.second) {
            for (int j = 0; j < numCols; ++j) {
                if (node.first < j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        } else if (node.first > node.second) {
            // Controllo per i > j
            for (int j = 0; j < numCols; ++j) {
                if (node.first > j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        } else {
            // Controllo per i == j
            for (int j = 0; j < numCols; ++j) {
                if (node.first == j && !std::isnan(denseMatrix[node.first * numCols + j])) {
                    int index = atomicAdd(nextFrontierSize, 1);
                    nextFrontier[index] = {node.first, j};
                    found = true;
                    break; // Interrompe il ciclo dopo aver trovato il primo elemento
                }
            }
        }

        if (found && node.first == source && node.second == source) {
            break; // Interrompe il ciclo se il nodo source è stato trovato
        }

        __syncthreads();

        if (tid == 0) {
            *currentFrontierSize = *nextFrontierSize;
            *nextFrontierSize = 0;
            Pair* temp = currentFrontier;
            currentFrontier = nextFrontier;
            nextFrontier = temp;
        }

        __syncthreads();

        if (*currentFrontierSize == 0) {
            break; // Interrompe il ciclo se non ci sono più nodi da esplorare
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cout << "Usage: ./exec matrix_file source\n";
        return 0;
    }

    std::string filename = argv[1];
    int source = atoi(argv[2]);

    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        return 1;
    }

    std::vector<std::vector<float>> denseMatrix;
    csr_to_dense(file, denseMatrix);
    std::cout << "Matrice formata\n";
    file.close();

    int numRows = denseMatrix.size();
    int numCols = denseMatrix[0].size();
    float* d_denseMatrix;
    hipMalloc(&d_denseMatrix, numRows * numCols * sizeof(float));
    hipMemcpy(d_denseMatrix, denseMatrix.data(), numRows * numCols * sizeof(float), hipMemcpyHostToDevice);

    // Inizializza currentFrontier con una singola coppia
    std::vector<Pair> hostCurrentFrontier = {{source, source}};
    Pair* d_currentFrontier;
    int* d_currentFrontierSize;
    hipMalloc(&d_currentFrontier, hostCurrentFrontier.size() * sizeof(Pair));
    hipMalloc(&d_currentFrontierSize, sizeof(int));
    hipMemcpy(d_currentFrontier, hostCurrentFrontier.data(), hostCurrentFrontier.size() * sizeof(Pair), hipMemcpyHostToDevice);
    int currentFrontierSize = hostCurrentFrontier.size();
    hipMemcpy(d_currentFrontierSize, &currentFrontierSize, sizeof(int), hipMemcpyHostToDevice);

    // Inizializza nextFrontier vuoto
    Pair* d_nextFrontier;
    int* d_nextFrontierSize;
    hipMalloc(&d_nextFrontier, numRows * numCols * sizeof(Pair)); // Assumiamo che la dimensione massima sia numRows * numCols
    hipMalloc(&d_nextFrontierSize, sizeof(int));
    int nextFrontierSize = 0;
    hipMemcpy(d_nextFrontierSize, &nextFrontierSize, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (currentFrontierSize + blockSize - 1) / blockSize;
    BFS_parallel<<<numBlocks, blockSize>>>(source, d_currentFrontier, d_currentFrontierSize, d_nextFrontier, d_nextFrontierSize, d_denseMatrix, numRows, numCols);

    // Copia i risultati dalla GPU all'host
    hipMemcpy(&nextFrontierSize, d_nextFrontierSize, sizeof(int), hipMemcpyDeviceToHost);
    std::vector<Pair> hostNextFrontier(nextFrontierSize);
    hipMemcpy(hostNextFrontier.data(), d_nextFrontier, nextFrontierSize * sizeof(Pair), hipMemcpyDeviceToHost);

    // Stampa i risultati
    for (const auto& p : hostNextFrontier) {
        std::cout << "Next Frontier Pair: (" << p.first << ", " << p.second << ")\n";
    }

    // Genera il file della matrice
    generate_matrix_file(denseMatrix, "output_matrix.txt");
    std::cout << "File della matrice generato\n";
    hipFree(d_denseMatrix);
    hipFree(d_currentFrontier);
    hipFree(d_currentFrontierSize);
    hipFree(d_nextFrontier);
    hipFree(d_nextFrontierSize);
    hipDeviceReset();

    return 0;
}
