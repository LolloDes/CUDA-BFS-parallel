
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>
#include <time.h>
#include <vector>
#include <sstream>
#include <algorithm>
#include <limits>
#include <numeric>
#include <cmath>  // Per std::isnan

#define MAX_FRONTIER_SIZE 128
struct Pair {
    int first;
    int second;
};
#define CHECK(call)                                                                 \
  {                                                                                 \
    const hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

#define CHECK_KERNELCALL()                                                          \
  {                                                                                 \
    const hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess) {                                                       \
      printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                           \
    }                                                                               \
  }

void csr_to_dense(std::ifstream &file, std::vector<std::vector<float>> &denseMatrix) {
    int numRows, numCols, numValues;
    file >> numRows >> numCols >> numValues;

    // Inizializza la matrice con NaN
    denseMatrix = std::vector<std::vector<float>>(numRows, std::vector<float>(numCols, std::numeric_limits<float>::quiet_NaN()));

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        row--; // Convert to 0-based index
        col--; // Convert to 0-based index
        denseMatrix[row][col] = value;
    }
}

void generate_matrix_file(const std::vector<std::vector<float>> &matrix, const std::string &filename) {
    std::ofstream matrixFile(filename);
    if (!matrixFile.is_open()) {
        std::cerr << "Cannot open matrix file!\n";
        return;
    }

    matrixFile << "Dense Matrix:" << std::endl;
    for (const auto &row : matrix) {
        for (const auto &val : row) {
            if (std::isnan(val)) {
                matrixFile << "- ";
            } else {
                matrixFile << val << " ";
            }
        }
        matrixFile << std::endl;
    }

    matrixFile.close();
}
__global__ void BFS_parallel(int source,std::vector<std::vector<float>> denseMatrix, Pair* currentfrontier, Pair* nextfrontier){
  /*

  1. leggere primo elemento
  2. controllare elementi vicini
  3. aggiungere al vettore livello
  while(true){
    Pair node = currentFrontier[tid];
    bool found=false;
    if (node.first < node.second) {
    // Controllo per i < j
      for (int j = 0; j < numCols; ++j) {
        if (node.first < j && !std::isnan(denseMatrix[node.first * numCols + j])) {
            int index = atomicAdd(nextFrontierSize, 1);
            nextFrontier[index] = {node.first, j};
            found = true;
            break; // Interrompe il ciclo dopo aver trovato il primo elemento
            }
        }
    } else if (node.first > node.second) {
    // Controllo per i > j
      for (int j = 0; j < numCols; ++j) {
        if (node.first > j && !std::isnan(denseMatrix[node.first * numCols + j])) {
          int index = atomicAdd(nextFrontierSize, 1);
          nextFrontier[index] = {node.first, j};
          found = true;
          break; // Interrompe il ciclo dopo aver trovato il primo elemento
          }
        }
      } else {
    // Controllo per i == j
      for (int j = 0; j < numCols; ++j) {
        if (node.first == j && !std::isnan(denseMatrix[node.first * numCols + j])) {
          int index = atomicAdd(nextFrontierSize, 1);
          nextFrontier[index] = {node.first, j};
          break; // Interrompe il ciclo dopo aver trovato il primo elemento
          }
        }
      }
  }
    4. ripetere in parallelo
      currentfrontier=nextfrontier;
      nextfrontier.clear();
    }
    if (found && node.first == source && node.second == source)
      break; // Interrompe il ciclo se il nodo source è stato trovato
  }
  5. vettore distanza
   5.1. col=row diminuisce ad aumentare di row e col
   5.2. controllo se NaN
   5.3. NaN->passo al prossimo
   5.4. Non NaN->
    5.4.1. scrivo numero nodo, 
    5.4.2. aumento distanza, 
    5.4.3. scrivo distanza
  6. stampa vettore distanze in un file*/
}
int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cout << "Usage: ./exec matrix_file source\n";
        return 0;
    }

    std::string filename = argv[1];
    int source = atoi(argv[2]);

    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "File cannot be opened!\n";
        return 1;
    }

    std::vector<std::vector<float>> denseMatrix;
    csr_to_dense(file, denseMatrix);
    file.close();

    generate_matrix_file(denseMatrix, "dense_matrix.txt");

    return 0;
}
